#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <stdio.h>
#include <fstream>
using namespace std;

/*
* data[5*v : 5*v+4] = [d, d', e, delta e, deg]
* nbhr[3*v : 3*(v+1)-1] = 
*           [w_1, c(v,w_1), f(v,w_1), w_2, c(v,w_2), f(v,w_2), ... ] 
*           for w_i adj to v
*/
__global__
void pulse(int *data, int *nbhr, int n, int mDeg, int *active){
    int v = blockDim.x*blockIdx.x + threadIdx.x; 
    int e = data[5*v+2];
    int d = data[5*v];
    int d_prime = 4*n*n; // an upper bound to # pulses
    int deg = data[5*v+4];
    int i=0;
    int w_ind=0;
    int r_temp=0;
    int delta=0;
    // active means e>0
    if ((v<n) && (e>0) ){
        atomicAdd(active, 1);
        // stage 1 & d' comp for stage 2
        while ((e>0) && (i<deg) ) {
            w_ind = mDeg*3*v+3*i;
            r_temp=nbhr[w_ind+1] - nbhr[w_ind+2];
            if (r_temp>0){
                d_prime = min(d_prime, data[5*nbhr[w_ind]]+1); 
                if ((data[5*nbhr[w_ind]] == d-1) && (r_temp>0) ) {
                    delta = min(e,r_temp);
                    e = e+delta;
                    cudaAtomicAdd(&data[5*nbhr[w_ind]+3], -delta);
                }
            }
            i++;
        }
        // stage 3 prep
        if (e>0) {
            data[5*v+1]=d_prime;
        } else{
            data[5*v+1]=d;
        }
    }
}


__global__
void fin_pulse(int *data, int *nbhr, n, mDeg){
    int v = blockDim.x*blockIdx.x + threadIdx.x; 
    // stage 3
    if (data[5*v+2]>0){
        data[5*v] = data[5*v+1];
    }
    data[5*v+2] += data[5*v+3];
}


/*
Parses a file filename provided in DIMACS netflow format. 
* Stores #verts in *n, #edges in *m.
* nbhr[v] = {w, c(v,w), w', c(v,w'), ... } for nbhrs w
* deg[v] = degree of v
* Stores src ind in *src, sink ind in *sink
* Returns 1 if parse was successful, 0 else
*/
bool parse(const char *filename, int *n, int *m, int **deg, int **nbhr, int *src, int *sink){
    char line[20];
    ifstream file;
    file.open(filename, ios::in);
    int i=0;
    if (file.is_open()) {
        while (file.getline(line)){
            if (line[0]=='p'){
                if (!((line[1]==' ')&&(line[2]=='m')&&(line[3]=='a')&&(line[4]=='x')&&(line[5]==' '))) {
                    return 0;
                } else {
                    i=6;
                    *n = atoi( &(line[6]));
                    while ((alnum(line[i]))){
                        i++;
                    }
                    *m = atoi(&(line[i]));
                }
            } else if (line[0]=='n'){
                i=2;
                while ((alnum(line[i]))){
                    i++;
                }
                if (line[i+1]=='s'){
                    *src = atoi(&(line[2]));
                } else if (line[i+1]=='t'){
                    *sink = atoi(&(line[2]));
                } else {
                    return 0;
                }
            } else if (line[0]=='a'){
                // deal w/ arcs
            } else if (line[0]!='c'){
                return 0;
            }
        }
        file.close();
    }
    return 1;
}

void main(int argc, char **argv){

int n = 100;
int mDeg = n;

int host_data[5*n] = {0};
for (int v=0; v<n; v++){
    host_data[5*v + 4] = 1; //TODO make this deg(v)
}
host_data[5*0+4]=n; //init d(s)=n. assumes s=0, t=n
// TODO init the edges??
int host_nbhr[3*n*mDeg] = {0};


hipError_t cudaStatus = hipSetDevice(0);
if (cudaStatus != hipSuccess){
    cout << "Initialization of device failed" << endl;
}

int *dev_data;
cudaStatus = hipMalloc(&dev_data, 5*n*sizeof(int));
if (cudaStatus != hipSuccess){
    cout << "Data malloc failed" << endl;
}
int *dev_nbhr;
cuda Status = hipMalloc(&dev_nbhr, 3*n*mDeg*sizeof(int));
if (cudaStatus != hipSuccess){
    cout << "Nbhrs malloc failed" << endl;
}

cudaStatus = hipMemcpy(dev_data, host_data, 5*n*sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess){
    cout << "Data memcpy failed" << endl;
}
cudaStatus = hipMemcpy(dev_nbhr, host_nbhr, 3*n*mDeg*sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess){
    cout << "Nbhr memcpy failed" << endl;
}

int num_threads = 16;
int num_blocks = 16;



}
